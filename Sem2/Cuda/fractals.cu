#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <GL/glut.h>
#define W 800
#define H 600
int max_steps = 10;
double left = -2.5;
double right = 1.5;
double top = 1.5;
double bottom = -1.5;

double get_time()
{
	double t ;
	//
	struct timeval* ptr = (struct timeval*)malloc( sizeof(struct timeval) ) ;
	//
	gettimeofday( ptr , NULL ) ; // second argument is time zone... NULL
	//
	t = ptr->tv_sec * 1000000.0 + ptr->tv_usec ;
	//
	free( ptr ) ;
	//
	return t / 1000000.0 ;
}

typedef struct {
    double real;
    double im;
} ImNum;

__device__ ImNum next_num(ImNum z, double a, double b){
    ImNum z_i = {pow(z.real, 2) - pow(z.im, 2) + a, 2*z.real*z.im + b};
    return z_i;
}
__global__ void run_pixel(double a, double hscale, double bottom, int max_steps, float *red, float *green, float *blue){
	int y = threadIdx.x;
	ImNum curr_z = {0, 0};
	double b = bottom + (y * hscale);
	int steps = 0;
	for(int i = 0; i < max_steps; i++){
		curr_z = next_num(curr_z, a, b);
		if(hypot(curr_z.real, curr_z.im) > 2){
			break;
		}
		steps++;
	}
	if(steps != max_steps){
		red[y] = (steps*1.0/max_steps);
		green[y] = 1-(steps*1.0/max_steps);
		blue[y] = (steps*1.0/max_steps);
	}
}

void displayfunc()
{
    double hscale = fabs(top-bottom)/H;
    double wscale = fabs(right-left)/W;
    glClear(GL_COLOR_BUFFER_BIT);
    double start = get_time();
    for(int x = 0; x < W; x++){
        double a = left + (x * wscale);
        float *red, *green, *blue;
        float *red_device, *green_device, *blue_device;
        red = (float*)malloc(H*sizeof(float));
        blue = (float*)malloc(H*sizeof(float));
        green = (float*)malloc(H*sizeof(float));
        hipMalloc((void**)&red_device, sizeof(float)*H);
        hipMalloc((void**)&blue_device, sizeof(float)*H);
        hipMalloc((void**)&green_device, sizeof(float)*H);
        for(int i = 0; i < H; i++){
			red[i] = 1;
			blue[i] = 1;
			green[i] = 1;
		}
		hipMemcpy(red_device, red, sizeof(float)*H, hipMemcpyHostToDevice);
		hipMemcpy(blue_device, blue, sizeof(float)*H, hipMemcpyHostToDevice);
		hipMemcpy(green_device, green, sizeof(float)*H, hipMemcpyHostToDevice);
		dim3 dimGrid(1), dimBlock(H);
		run_pixel<<<dimGrid,dimBlock>>>(a, hscale, bottom, max_steps, red_device, green_device, blue_device);
		hipMemcpy(red, red_device, sizeof(float)*H, hipMemcpyDeviceToHost);
		hipMemcpy(blue, blue_device, sizeof(float)*H, hipMemcpyDeviceToHost);
		hipMemcpy(green, green_device, sizeof(float)*H, hipMemcpyDeviceToHost);
		for(int y = 0; y < H; y++){
			glColor3f(red[y], green[y], blue[y]);
			glBegin(GL_POINTS);
			glVertex2f(x,y);
			glEnd();
		}
		
		free(red);
		free(blue);
		free(green);
		hipFree(red_device);
		hipFree(blue_device);
		hipFree(green_device);
    }
    glutSwapBuffers();
    double end = get_time();
    printf("dx: %f\ndy: %f\n", fabs(right-left), fabs(top-bottom));
    printf("max steps: %d\n", max_steps);
    printf("time: %f\n", end - start);
    printf("***********************************\n");

}
void reshapefunc(int wscr,int hscr)
{
   glViewport(0,0,(GLsizei)W,(GLsizei)H);
   glMatrixMode(GL_PROJECTION);
   glLoadIdentity();
   gluOrtho2D(0.0,1.0*W,0.0,1.0*H); // always a square
   glMatrixMode(GL_MODELVIEW);
}
void mousefunc(int button,int state,int xscr,int yscr)
{
    if(button == 3 && state == GLUT_DOWN){
        max_steps *= 2;
    }
    if(button == 4 && state == GLUT_DOWN){
        max_steps /= 2;
    }
    if(button == GLUT_LEFT_BUTTON && state == GLUT_DOWN){
        double hscale = fabs(top-bottom)/H;
        double wscale = fabs(right-left)/W;
        double x_coord = left + (xscr * wscale);
        double y_coord = bottom + ((H-yscr) * hscale);
        double temp_left = x_coord - (fabs(right-left)/4);
        double temp_right = x_coord + (fabs(right-left)/4);
        left = temp_left;
        right = temp_right;
        double temp_bottom = y_coord - fabs(top-bottom)/4;
        double temp_top = y_coord + fabs(top-bottom)/4;
        bottom = temp_bottom;
        top = temp_top;
    }
    if(button == GLUT_RIGHT_BUTTON && state == GLUT_DOWN){
        double hscale = fabs(top-bottom)/H;
        double wscale = fabs(right-left)/W;
        double x_coord = left + (xscr * wscale);
        double y_coord = bottom + ((H-yscr) * hscale);
        double temp_left = x_coord - (fabs(right-left));
        double temp_right = x_coord + (fabs(right-left));
        left = temp_left;
        right = temp_right;
        double temp_bottom = y_coord - fabs(top-bottom);
        double temp_top = y_coord + fabs(top-bottom);
        bottom = temp_bottom;
        top = temp_top;
    }
    glutPostRedisplay();
}
void keyfunc(unsigned char key,int xscr,int yscr)
{
    double hscale = fabs(top-bottom)/H;
    double wscale = fabs(right-left)/W;
    if(key == 'a'){
        left -= (W/10)*wscale;
        right -= (W/10)*wscale;
    }
    if(key == 'd'){
        left += (W/10)*wscale;
        right += (W/10)*wscale;
    }
    if(key == 'w'){
        bottom += (H/10)*hscale;
        top += (H/10)*hscale;
    }
    if(key == 's'){
        bottom -= (H/10)*hscale;
        top -= (H/10)*hscale;
    }
    glutPostRedisplay();
}
int main(int argc,char* argv[])
{
   glutInit(&argc,argv);
   glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
   glutInitWindowSize(W, H);
   glutInitWindowPosition(100,50);
   glutCreateWindow("Fractals Lab");
   glClearColor(1.0,1.0,1.0,0.0);
   glShadeModel(GL_SMOOTH);
   glutDisplayFunc(displayfunc);
   glutReshapeFunc(reshapefunc);
   glutMouseFunc(mousefunc);
   glutKeyboardFunc(keyfunc);
   glutMainLoop();
   return 0;
}

