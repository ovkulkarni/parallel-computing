//
// http://forums.nvidia.com/index.php?showtopic=34309
//

#include <hip/hip_runtime.h>
#include <stdio.h>

// called from host, run on device
__global__ void add_arrays_gpu(float *in1,float *in2,float *out)
{
	int idx=threadIdx.x; // flat model
	
	out[idx]=in1[idx]+in2[idx];
}
int main()
{
    hipDeviceProp_t c;
    hipGetDeviceProperties(&c, 0);
    printf("%s\n", c.name);
    printf("%d\n", c.maxThreadsPerBlock);
    printf("%d, %d, %d\n", c.maxThreadsDim[0], c.maxThreadsDim[1], c.maxThreadsDim[2]);
    printf("%d, %d, %d\n", c.maxGridSize[0], c.maxGridSize[1], c.maxGridSize[2]);
}
